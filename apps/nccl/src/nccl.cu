#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <algorithm>
#include <mscclpp/concurrency_device.hpp>
#include <mscclpp/core.hpp>
#include <mscclpp/sm_channel.hpp>
#include <mscclpp/sm_channel_device.hpp>
#include <unordered_map>
#include <vector>

#include "nccl.h"

#define NCCL_API extern "C" __attribute__((visibility("default")))

#define CUDACHECK(cmd)                                                                      \
  do {                                                                                      \
    hipError_t e = cmd;                                                                    \
    if (e != hipSuccess) {                                                                 \
      printf("Failed: Cuda error %s:%d '%s'\n", __FILE__, __LINE__, hipGetErrorString(e)); \
      exit(EXIT_FAILURE);                                                                   \
    }                                                                                       \
  } while (0)

#define NUM_CHANNELS_PER_CONNECTION 32

#if defined(__HIP_PLATFORM_AMD__)
#define WARP_SIZE 64
#else
#define WARP_SIZE 32
#endif

template <typename To, typename From>
__forceinline__ __device__ To bit_cast(const From& src) {
  static_assert(sizeof(To) == sizeof(From), "Size mismatch for bit_cast");

  union {
    From f;
    To t;
  } u;
  u.f = src;
  return u.t;
}

template <typename T>
__forceinline__ __device__ T add_elements(T a, T b) {
  return a + b;
}

template <>
__forceinline__ __device__ __half2 add_elements(__half2 a, __half2 b) {
  return __hadd2(a, b);
}

template <typename T>
__forceinline__ __device__ int4 add_vectors_helper(int4 a, int4 b) {
  int4 ret;
  ret.w = bit_cast<int, T>(add_elements(bit_cast<T, int>(a.w), bit_cast<T, int>(b.w)));
  ret.x = bit_cast<int, T>(add_elements(bit_cast<T, int>(a.x), bit_cast<T, int>(b.x)));
  ret.y = bit_cast<int, T>(add_elements(bit_cast<T, int>(a.y), bit_cast<T, int>(b.y)));
  ret.z = bit_cast<int, T>(add_elements(bit_cast<T, int>(a.z), bit_cast<T, int>(b.z)));
  return ret;
}

template <typename T>
__forceinline__ __device__ int4 add_vectors(int4 a, int4 b) {
  return add_vectors_helper<T>(a, b);
}

template <>
__forceinline__ __device__ int4 add_vectors<__half>(int4 a, int4 b) {
  return add_vectors_helper<__half2>(a, b);
}

template <typename T>
__forceinline__ __device__ uint2 add_vectors_helper(uint2 a, uint2 b) {
  uint2 ret;
  ret.x = bit_cast<int, T>(add_elements(bit_cast<T, int>(a.x), bit_cast<T, int>(b.x)));
  ret.y = bit_cast<int, T>(add_elements(bit_cast<T, int>(a.y), bit_cast<T, int>(b.y)));
  return ret;
}

template <typename T>
__forceinline__ __device__ uint2 add_vectors(uint2 a, uint2 b) {
  return add_vectors_helper<T>(a, b);
}

template <>
__forceinline__ __device__ uint2 add_vectors<__half>(uint2 a, uint2 b) {
  return add_vectors_helper<__half2>(a, b);
}

template <typename T>
__forceinline__ __device__ int add_vectors_helper(int a, int b) {
  return bit_cast<int, T>(add_elements(bit_cast<T, int>(a), bit_cast<T, int>(b)));
}

template <typename T>
__forceinline__ __device__ int add_vectors(int a, int b) {
  return add_vectors_helper<T>(a, b);
}

template <>
__forceinline__ __device__ int add_vectors<__half>(int a, int b) {
  return add_vectors_helper<__half2>(a, b);
}

template <typename T>
__forceinline__ __device__ uint32_t add_vectors_helper(uint32_t a, uint32_t b) {
  return bit_cast<uint32_t, T>(add_elements(bit_cast<T, uint32_t>(a), bit_cast<T, uint32_t>(b)));
}

template <typename T>
__forceinline__ __device__ uint32_t add_vectors(uint32_t a, uint32_t b) {
  return add_vectors_helper<T>(a, b);
}

template <>
__forceinline__ __device__ uint32_t add_vectors<__half>(uint32_t a, uint32_t b) {
  return add_vectors_helper<__half2>(a, b);
}

template <typename T>
__forceinline__ __device__ void vectorSum(T* dst, T* src, size_t nElem, int blockId, int nBlocks) {
  size_t nInt4 = nElem / 4;
  size_t nLastInts = nElem % 4;
  int4* dst4 = (int4*)dst;
  int4* src4 = (int4*)src;
  for (size_t i = threadIdx.x + blockId * blockDim.x; i < nInt4; i += blockDim.x * nBlocks) {
    dst4[i] = add_vectors<T>(dst4[i], src4[i]);
  }
  if (nLastInts > 0) {
    int* dstLast = ((int*)dst) + nInt4 * 4;
    int* srcLast = ((int*)src) + nInt4 * 4;
    for (size_t i = threadIdx.x + blockId * blockDim.x; i < nLastInts; i += blockDim.x * nBlocks) {
      dstLast[i] = add_vectors<T>(dstLast[i], srcLast[i]);
    }
  }
}

template <typename T>
__forceinline__ __device__ void vectorSum(T* dst, T* src, size_t nElem) {
  vectorSum(dst, src, nElem, blockIdx.x, gridDim.x);
}

// TODO:
static const int nRanksPerNode = 8;
// Only use scratch buffer for message size less then 1MB
static const int scratchSize = 1024 * 1024 * 8;

// static const mscclpp::Transport IBs[] = {mscclpp::Transport::IB0, mscclpp::Transport::IB1, mscclpp::Transport::IB2,
//                             mscclpp::Transport::IB3, mscclpp::Transport::IB4, mscclpp::Transport::IB5,
//                             mscclpp::Transport::IB6, mscclpp::Transport::IB7};

__constant__ mscclpp::DeviceHandle<mscclpp::SmChannel> constSmChannels[256];
__constant__ mscclpp::DeviceHandle<mscclpp::SmChannel> constSmOutChannels[256];
__device__ mscclpp::DeviceSyncer deviceSyncer;

struct channelKey {
  const void* sendbuff;
  const void* recvbuff;
  size_t bytes;
  bool operator==(const channelKey& other) const {
    return sendbuff == other.sendbuff && recvbuff == other.recvbuff && bytes == other.bytes;
  }
};

namespace std {
template <>
struct hash<channelKey> {
  std::size_t operator()(const channelKey& k) const {
    return std::hash<const void*>()(k.sendbuff) ^ std::hash<const void*>()(k.recvbuff) ^ std::hash<size_t>()(k.bytes);
  }
};
}  // namespace std

struct ChannelInfo {
  std::vector<mscclpp::SmChannel> smChannels;
  std::vector<mscclpp::SmChannel> smOutChannels;
  /*std::vector<mscclpp::DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles;
  std::vector<mscclpp::DeviceHandle<mscclpp::SmChannel>> smOutChannelDeviceHandles;*/
  void* smChannelDeviceHandles;
  void* smOutChannelDeviceHandles;  
};

struct ncclComm {
  std::shared_ptr<mscclpp::Communicator> comm;
  std::vector<std::shared_ptr<mscclpp::Connection>> connections;
  std::vector<std::shared_ptr<mscclpp::SmDevice2DeviceSemaphore>> smSemaphores;

  std::unordered_map<channelKey, ChannelInfo> channelInfos;
  std::shared_ptr<char> scratchBuff;
  std::vector<mscclpp::RegisteredMemory> remoteScratchRegMemories;
};

hipError_t allreduce(int* buff, int* scratch, void* resultBuff, int rank, int nRanksPerNode, int worldSize,
                      size_t nelems, hipStream_t stream);

#include <mscclpp/packet_device.hpp>
#include <mscclpp/sm_channel_device.hpp>

// extern __constant__ mscclpp::SmChannelDeviceHandle *constSmChannels;
__device__ uint64_t globalFlag;

template <typename T>
__global__ void allreduce6(T* buff, T* scratch, T* resultBuff, int rank, int nRanksPerNode, int worldSize,
                           size_t nelems) {
  // This version of allreduce only works for single nodes
  if (worldSize != nRanksPerNode) return;
  nelems = nelems / (sizeof(int) / sizeof(T));
  const int nPeers = nRanksPerNode - 1;
  const int nPkts = nelems / 2;
  const int nelemsPerRank = nelems / worldSize;
  const int nPktsPerRank = nelemsPerRank / 2;
  // flag for packets. Initially 1
  const uint32_t flag = (uint32_t)globalFlag + 1;
  // thread block & channel info
  const int nBlocksPerPeer = gridDim.x / nPeers;
  const int localBlockIdx = blockIdx.x % nBlocksPerPeer;
  const int peerIdx = blockIdx.x / nBlocksPerPeer;
  const int remoteRank = peerIdx < rank ? peerIdx : peerIdx + 1;
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    globalFlag += 1;
  }
}

template <typename T>
__global__ void allreduce1(T* src, T* dst, int rank, int nranks, size_t nelems) {
  const size_t chunkSize = nelems / nranks;
  if (nranks == 1) return;
  const int nPeer = nranks - 1;
  const size_t indexOffset = rank * chunkSize;
  const size_t vectorSize = sizeof(int4) / sizeof(T);
  const size_t indexOffset4 = indexOffset / vectorSize;
  int4* src4 = (int4*)src;
  int4* dst4 = (int4*)dst;
  const int tid = threadIdx.x + blockIdx.x * blockDim.x;

}

template <typename T>
__global__ void allreduce7(T* buff, T* scratch, T* resultBuff, int rank, int nRanksPerNode, int worldSize,
                           size_t nelems) {
  // This version of allreduce only works for single nodes
  if (worldSize != nRanksPerNode) return;
  nelems = nelems / (sizeof(int) / sizeof(T));
  if (threadIdx.x == 0 && blockIdx.x == 0) {
    globalFlag += 1;
  }
}

template <typename T>
hipError_t allreduce(T* buff, T* scratch, T* resultBuff, int rank, int nRanksPerNode, int worldSize, size_t nelems,
                      hipStream_t stream) {
  if (sizeof(T) * nelems <= (1 << 20)) {
#if defined(__HIP_PLATFORM_AMD__)
    int nBlocks = 28;
    int nThreadsPerBlock = 1024;
    if (nelems >= 8192) {
      nBlocks = 56;
      nThreadsPerBlock = (nelems <= 76800) ? 512 : 1024;
    }
    allreduce7<<<nBlocks, nThreadsPerBlock, 0, stream>>>(buff, scratch, resultBuff, rank, nRanksPerNode, worldSize,
                                                         nelems);
#else
    allreduce6<<<21, 512, 0, stream>>>(buff, scratch, resultBuff, rank, nRanksPerNode, worldSize, nelems);
#endif
  } else {
    allreduce1<<<24, 1024, 0, stream>>>(buff, resultBuff, rank, worldSize, nelems);
  }
  return hipGetLastError();
}

__global__ void __launch_bounds__(1024, 1)
    allgather5(size_t rank, [[maybe_unused]] size_t worldSize, size_t nRanksPerNode, size_t nelemsPerGPU, void* smChannel) {
  const size_t nBlock = gridDim.x;
  if (blockIdx.x >= nBlock) return;

  const size_t tid = threadIdx.x + blockIdx.x * blockDim.x;
  const size_t lid = tid % WARP_SIZE;
  const size_t wid = tid / WARP_SIZE;

  const size_t nThread = blockDim.x * nBlock;
  const size_t nWarp = nThread / WARP_SIZE;
  const size_t nPeer = nRanksPerNode - 1;
  const size_t chanOffset = nPeer * blockIdx.x;
  mscclpp::DeviceHandle<mscclpp::SmChannel> *constSmChannels =  (mscclpp::DeviceHandle<mscclpp::SmChannel>*) smChannel;
  auto smChans = constSmChannels + chanOffset;

  if (wid < nPeer && lid == 0) {
    smChans[wid].relaxedSignal();
    smChans[wid].wait();
  }
  __syncthreads();
  const size_t bytesPerGPU = nelemsPerGPU * sizeof(int);
  const size_t bytes = bytesPerGPU * nPeer;
  size_t unitBytesPerThread;
  if (bytes >= nThread * 64) {
    unitBytesPerThread = 64;
  } else {
    unitBytesPerThread = 16;
  }
  const size_t unitBytesPerWarp = unitBytesPerThread * WARP_SIZE;
  const size_t unitBytes = unitBytesPerWarp * nWarp;
  const size_t nLoop = bytes / unitBytes;

  if (nLoop > 0) {
    // First loop unrolling
    const size_t peerIdx = wid % nPeer;
    const size_t remoteRankLocalIndex = (peerIdx < rank ? peerIdx : peerIdx + 1);
    const size_t offset = bytesPerGPU * remoteRankLocalIndex + (wid / nPeer) * unitBytesPerWarp;
    smChans[peerIdx].get<16, false>(offset, unitBytesPerWarp, lid, WARP_SIZE);
  }

  for (size_t i = 1; i < nLoop; ++i) {
    const size_t gWid = wid + i * nWarp;
    const size_t peerIdx = gWid % nPeer;
    const size_t remoteRankLocalIndex = (peerIdx < rank ? peerIdx : peerIdx + 1);
    const size_t offset = bytesPerGPU * remoteRankLocalIndex + (gWid / nPeer) * unitBytesPerWarp;
    smChans[peerIdx].get<16, false>(offset, unitBytesPerWarp, lid, WARP_SIZE);
  }

  if (bytes % unitBytes > 0) {
    const size_t gWid = wid + nLoop * nWarp;
    const size_t peerIdx = gWid % nPeer;
    const size_t remoteRankLocalIndex = (peerIdx < rank ? peerIdx : peerIdx + 1);
    const size_t offsetWithinRank = (gWid / nPeer) * unitBytesPerWarp;
    const size_t offset = bytesPerGPU * remoteRankLocalIndex + offsetWithinRank;
    const size_t remainBytes = (offsetWithinRank + unitBytesPerWarp > bytesPerGPU)
                                   ? ((bytesPerGPU > offsetWithinRank) ? (bytesPerGPU - offsetWithinRank) : 0)
                                   : unitBytesPerWarp;
    if (remainBytes > 0) {
      smChans[peerIdx].get<16, true>(offset, remainBytes, lid, WARP_SIZE);
    }
  }
}

template <typename T>
hipError_t allgather(T* buff, T* scratch, T* resultBuff, int rank, int nRanksPerNode, int worldSize, size_t nelems,
                      hipStream_t stream, void* smChannel) {
  hipError_t err = hipMemcpyAsync(resultBuff + nelems * rank, buff, nelems * sizeof(T), hipMemcpyDeviceToDevice, stream);
  if (err != hipSuccess) return err;
  allgather5<<<24, 1024, 0, stream>>>(rank, worldSize, nRanksPerNode, nelems, smChannel);
  return hipGetLastError();
}

static size_t ncclTypeSize(ncclDataType_t type) {
  switch (type) {
    case ncclInt8:
    case ncclUint8:
      return 1;
    case ncclFloat16:
      return 2;
    case ncclInt32:
    case ncclUint32:
      return 4;
    case ncclInt64:
    case ncclUint64:
      return 8;
    case ncclFloat32:
      return 4;
    case ncclFloat64:
      return 8;
#if defined(__CUDA_BF16_TYPES_EXIST__)
    case ncclBfloat16:
      return 2;
#endif  // defined(__CUDA_BF16_TYPES_EXIST__)
#if defined(__CUDA_FP8_TYPES_EXIST__)
    case ncclFp8E4M3:
    case ncclFp8E5M2:
      return 1;
#endif  // defined(__CUDA_FP8_TYPES_EXIST__)
    case ncclNumTypes:
      return 0;
  }
  return 0;
}

static mscclpp::Transport getTransport(int rank, int peerRank) {
  // if (rank / nRanksPerNode == peerRank / nRanksPerNode) {
  //   return mscclpp::Transport::CudaIpc;
  // } else {
  //   return IBs[rank % nRanksPerNode];
  // }
  return mscclpp::Transport::CudaIpc;
}

static std::vector<mscclpp::RegisteredMemory> setupRemoteMemories(std::shared_ptr<mscclpp::Communicator> comm, int rank,
                                                                  void* buff, size_t bytes,
                                                                  mscclpp::TransportFlags transport) {
  std::vector<mscclpp::RegisteredMemory> remoteMemories;
  mscclpp::RegisteredMemory memory = comm->registerMemory(buff, bytes, transport);
  std::vector<mscclpp::NonblockingFuture<mscclpp::RegisteredMemory>> remoteRegMemoryFutures;
  for (int i = 0; i < comm->bootstrap()->getNranks(); i++) {
    if (i == rank) continue;
    remoteRegMemoryFutures.push_back(comm->recvMemoryOnSetup(i, 0));
    comm->sendMemoryOnSetup(memory, i, 0);
  }
  comm->setup();
  std::transform(remoteRegMemoryFutures.begin(), remoteRegMemoryFutures.end(), std::back_inserter(remoteMemories),
                 [](const auto& future) { return future.get(); });
  return remoteMemories;
}

static std::vector<mscclpp::SmChannel> setupSmChannels(ncclComm_t comm,
                                                       const std::vector<mscclpp::RegisteredMemory>& remoteMemories,
                                                       void* src) {
  std::vector<mscclpp::SmChannel> channels;
  std::vector<std::shared_ptr<mscclpp::SmDevice2DeviceSemaphore>>& smSemaphores = comm->smSemaphores;
  size_t nConnections = comm->connections.size();
  for (size_t idx = 0; idx < NUM_CHANNELS_PER_CONNECTION; ++idx) {
    for (size_t cid = 0; cid < nConnections; ++cid) {
      if (comm->connections[cid]->transport() == mscclpp::Transport::CudaIpc) {
        channels.emplace_back(smSemaphores[idx * nConnections + cid], remoteMemories[cid], src, nullptr);
      }
    }
  }
  return channels;
}

NCCL_API ncclResult_t ncclGetVersion(int* version) {
  if (version == nullptr) return ncclInvalidArgument;
  *version = MSCCLPP_VERSION;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclGetUniqueId(ncclUniqueId* uniqueId) {
  if (uniqueId == nullptr) return ncclInvalidArgument;
  if (MSCCLPP_UNIQUE_ID_BYTES != NCCL_UNIQUE_ID_BYTES) return ncclInternalError;
  mscclpp::UniqueId id = mscclpp::TcpBootstrap::createUniqueId();
  memcpy(uniqueId, &id, sizeof(ncclUniqueId));
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommInitRankConfig(ncclComm_t* comm, int nranks, ncclUniqueId commId, int rank,
                                             ncclConfig_t* config) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclCommInitRank(ncclComm_t* comm, int nranks, ncclUniqueId commId, int rank) {
  if (comm == nullptr) return ncclInvalidArgument;
  if (nranks < 0 || rank < 0 || rank >= nranks) return ncclInvalidArgument;
  std::shared_ptr<mscclpp::TcpBootstrap> bootstrap = std::make_shared<mscclpp::TcpBootstrap>(rank, nranks);
  mscclpp::UniqueId id;
  memcpy(id.data(), &commId, sizeof(ncclUniqueId));
  bootstrap->initialize(id);
  std::shared_ptr<mscclpp::Communicator> mscclppComm = std::make_shared<mscclpp::Communicator>(bootstrap);
  std::vector<mscclpp::NonblockingFuture<std::shared_ptr<mscclpp::Connection>>> connectionFutures;

  for (int i = 0; i < mscclppComm->bootstrap()->getNranks(); i++) {
    if (i == rank) continue;
    mscclpp::Transport transport = getTransport(rank, i);
    connectionFutures.push_back(mscclppComm->connectOnSetup(i, 0, transport));
  }
  mscclppComm->setup();

  std::vector<std::shared_ptr<mscclpp::Connection>> connections;
  std::transform(connectionFutures.begin(), connectionFutures.end(), std::back_inserter(connections),
                 [](const auto& future) { return future.get(); });

  std::vector<std::shared_ptr<mscclpp::SmDevice2DeviceSemaphore>> smSemaphores;
  for (size_t idx = 0; idx < NUM_CHANNELS_PER_CONNECTION; ++idx) {
    for (size_t cid = 0; cid < connections.size(); ++cid) {
      if (connections[cid]->transport() == mscclpp::Transport::CudaIpc) {
        smSemaphores.emplace_back(
            std::make_shared<mscclpp::SmDevice2DeviceSemaphore>(*(mscclppComm), connections[cid]));
      }
    }
  }
  mscclppComm->setup();

  ncclComm* commPtr = new ncclComm();
  commPtr->comm = mscclppComm;
  commPtr->connections = std::move(connections);
  commPtr->smSemaphores = std::move(smSemaphores);
  // using scratch buffer for message size less then 1MB
  commPtr->scratchBuff = mscclpp::allocExtSharedCuda<char>(scratchSize);
  commPtr->remoteScratchRegMemories =
      setupRemoteMemories(commPtr->comm, rank, commPtr->scratchBuff.get(), scratchSize, mscclpp::Transport::CudaIpc);

  *comm = commPtr;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommInitAll(ncclComm_t* comm, int ndev, const int* devlist) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclCommFinalize(ncclComm_t comm) {
  comm->comm->bootstrap()->barrier();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommDestroy(ncclComm_t comm) {
  if (comm == nullptr) return ncclInvalidArgument;
  delete comm;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommAbort(ncclComm_t comm) {
  // TODO: implement this function
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommSplit(ncclComm_t comm, int color, int key, ncclComm_t* newcomm, ncclConfig_t* config) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API const char* ncclGetErrorString(ncclResult_t result) {
  switch (result) {
    case ncclSuccess:
      return "no error";
    case ncclUnhandledCudaError:
      return "unhandled cuda error (run with NCCL_DEBUG=INFO for details)";
    case ncclSystemError:
      return "unhandled system error (run with NCCL_DEBUG=INFO for details)";
    case ncclInternalError:
      return "internal error - please report this issue to the NCCL developers";
    case ncclInvalidArgument:
      return "invalid argument (run with NCCL_DEBUG=WARN for details)";
    case ncclInvalidUsage:
      return "invalid usage (run with NCCL_DEBUG=WARN for details)";
    case ncclRemoteError:
      return "remote process exited or there was a network error";
    case ncclInProgress:
      return "NCCL operation in progress";
    default:
      return "unknown result code";
  }
}

NCCL_API const char* ncclGetLastError(ncclComm_t comm) {
  // TODO: implement this function
  return nullptr;
}

NCCL_API ncclResult_t ncclCommGetAsyncError(ncclComm_t comm, ncclResult_t* asyncError) {
  if (asyncError == nullptr) return ncclInvalidArgument;
  *asyncError = ncclSuccess;
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommCount(const ncclComm_t comm, int* count) {
  if (comm == nullptr || count == nullptr) return ncclInvalidArgument;
  *count = comm->comm->bootstrap()->getNranks();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommCuDevice(const ncclComm_t comm, int* device) {
  if (comm == nullptr || device == nullptr) return ncclInvalidArgument;
  *device = comm->comm->bootstrap()->getRank();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclCommUserRank(const ncclComm_t comm, int* rank) {
  if (comm == nullptr || rank == nullptr) return ncclInvalidArgument;
  *rank = comm->comm->bootstrap()->getRank();
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclRedOpCreatePreMulSum(ncclRedOp_t* op, void* scalar, ncclDataType_t datatype,
                                               ncclScalarResidence_t residence, ncclComm_t comm) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclRedOpDestroy(ncclRedOp_t op, ncclComm_t comm) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclReduce(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                 ncclRedOp_t op, int root, ncclComm_t comm, hipStream_t stream) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclBcast(void* buff, size_t count, ncclDataType_t datatype, int root, ncclComm_t comm,
                                hipStream_t stream) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclBroadcast(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                    int root, ncclComm_t comm, hipStream_t stream) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclAllReduce(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                    ncclRedOp_t op, ncclComm_t comm, hipStream_t stream) {
  size_t bytes = count * ncclTypeSize(datatype);
  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr) return ncclInvalidArgument;
  int rank = comm->comm->bootstrap()->getRank();
  channelKey key{sendbuff, recvbuff, bytes};
  /*if (bytes <= 1 << 20) {
    auto it = comm->channelInfos.find(key);
    if (it == comm->channelInfos.end()) {
      std::vector<mscclpp::SmChannel> channels =
          setupSmChannels(comm, comm->remoteScratchRegMemories, const_cast<void*>(sendbuff));
      std::vector<mscclpp::DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles;
      std::transform(channels.begin(), channels.end(), std::back_inserter(smChannelDeviceHandles),
                     [](const mscclpp::SmChannel& smChannel) { return mscclpp::deviceHandle(smChannel); });
      ChannelInfo channelInfo{channels, {}, smChannelDeviceHandles, {}};
      it = comm->channelInfos.emplace(key, channelInfo).first;
    }
    CUDACHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(
        constSmChannels), it->second.smChannelDeviceHandles.data(),
        sizeof(mscclpp::DeviceHandle<mscclpp::SmChannel>) * it->second.smChannelDeviceHandles.size(), 0,
        hipMemcpyHostToDevice, stream));
  } else {
    auto it = comm->channelInfos.find(key);
    if (it == comm->channelInfos.end()) {
      std::vector<mscclpp::RegisteredMemory> remoteMemories =
          setupRemoteMemories(comm->comm, rank, const_cast<void*>(sendbuff), bytes, mscclpp::Transport::CudaIpc);
      std::vector<mscclpp::SmChannel> channels = setupSmChannels(comm, remoteMemories, const_cast<void*>(sendbuff));
      std::vector<mscclpp::DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles;
      std::transform(channels.begin(), channels.end(), std::back_inserter(smChannelDeviceHandles),
                     [](const mscclpp::SmChannel& smChannel) { return mscclpp::deviceHandle(smChannel); });
      ChannelInfo channelInfo{channels, {}, smChannelDeviceHandles, {}};
      it = comm->channelInfos.emplace(key, channelInfo).first;
      if (sendbuff != recvbuff) {
        std::vector<mscclpp::RegisteredMemory> remoteMemories =
            setupRemoteMemories(comm->comm, rank, recvbuff, bytes, mscclpp::Transport::CudaIpc);
        std::vector<mscclpp::SmChannel> outChannels = setupSmChannels(comm, remoteMemories, recvbuff);
        it->second.smOutChannels = outChannels;
        std::transform(outChannels.begin(), outChannels.end(), std::back_inserter(it->second.smOutChannelDeviceHandles),
                       [](const mscclpp::SmChannel& smChannel) { return mscclpp::deviceHandle(smChannel); });
      }
    }
    CUDACHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(
        constSmChannels), it->second.smChannelDeviceHandles.data(),
        sizeof(mscclpp::DeviceHandle<mscclpp::SmChannel>) * it->second.smChannelDeviceHandles.size(), 0,
        hipMemcpyHostToDevice, stream));
    if (sendbuff != recvbuff) {
      CUDACHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(
          constSmOutChannels), it->second.smOutChannelDeviceHandles.data(),
          sizeof(mscclpp::DeviceHandle<mscclpp::SmChannel>) * it->second.smOutChannelDeviceHandles.size(), 0,
          hipMemcpyHostToDevice, stream));
    } else {
      CUDACHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(
          constSmOutChannels), it->second.smChannelDeviceHandles.data(),
          sizeof(mscclpp::DeviceHandle<mscclpp::SmChannel>) * it->second.smChannelDeviceHandles.size(), 0,
          hipMemcpyHostToDevice, stream));
    }
  }

  switch (datatype) {
    case ncclFloat16:
      CUDACHECK(allreduce((half*)sendbuff, (half*)comm->scratchBuff.get(), (half*)recvbuff, rank, nRanksPerNode,
                          comm->comm->bootstrap()->getNranks(), count, stream));
      break;
    case ncclFloat32:
      CUDACHECK(allreduce((float*)sendbuff, (float*)comm->scratchBuff.get(), (float*)recvbuff,
                          comm->comm->bootstrap()->getRank(), nRanksPerNode, comm->comm->bootstrap()->getNranks(),
                          count, stream));
      break;
    case ncclInt32:
    case ncclUint32:
      CUDACHECK(allreduce((int*)sendbuff, (int*)comm->scratchBuff.get(), (int*)recvbuff,
                          comm->comm->bootstrap()->getRank(), nRanksPerNode, comm->comm->bootstrap()->getNranks(),
                          count, stream));
      break;
    default:
      return ncclInvalidArgument;
  }*/
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclReduceScatter(const void* sendbuff, void* recvbuff, size_t recvcount, ncclDataType_t datatype,
                                        ncclRedOp_t op, ncclComm_t comm, hipStream_t stream) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclAllGather(const void* sendbuff, void* recvbuff, size_t sendcount, ncclDataType_t datatype,
                                    ncclComm_t comm, hipStream_t stream) {
  size_t bytes = sendcount * ncclTypeSize(datatype);
  if (sendbuff == nullptr || recvbuff == nullptr || bytes == 0 || comm == nullptr) return ncclInvalidArgument;
  int rank = comm->comm->bootstrap()->getRank();
  int nRank = comm->comm->bootstrap()->getNranks();
  channelKey key{sendbuff, recvbuff, bytes};

  mscclpp::DeviceHandle<mscclpp::SmChannel> *constSmChannels;
  mscclpp::DeviceHandle<mscclpp::SmChannel> *constSmOutChannels;

  auto it = comm->channelInfos.find(key);
  if (it == comm->channelInfos.end()) {
    std::vector<mscclpp::RegisteredMemory> remoteMemories =
        setupRemoteMemories(comm->comm, rank, const_cast<void*>(recvbuff), bytes * nRank,
                            mscclpp::Transport::CudaIpc);
    std::vector<mscclpp::SmChannel> channels =
        setupSmChannels(comm, remoteMemories, const_cast<void*>(recvbuff));
    std::vector<mscclpp::DeviceHandle<mscclpp::SmChannel>> smChannelDeviceHandles;
    std::transform(channels.begin(), channels.end(), std::back_inserter(smChannelDeviceHandles),
                   [](const mscclpp::SmChannel& smChannel) { return mscclpp::deviceHandle(smChannel); });
  
    mscclpp::AvoidCudaGraphCaptureGuard cgcGuard;
    hipMalloc((void**)&constSmChannels, sizeof(mscclpp::DeviceHandle<mscclpp::SmChannel>) * smChannelDeviceHandles.size());
    hipMemcpy(constSmChannels, smChannelDeviceHandles.data(), sizeof(mscclpp::DeviceHandle<mscclpp::SmChannel>) * smChannelDeviceHandles.size(), hipMemcpyHostToDevice);
    ChannelInfo channelInfo{channels, {}, constSmChannels, {}};
    it = comm->channelInfos.emplace(key, channelInfo).first;

  } else {
	constSmChannels = (mscclpp::DeviceHandle<mscclpp::SmChannel> *) it->second.smChannelDeviceHandles;	
  }

  // TODO: if sendbuff and recvbuff don't change, we can avoid copying smChannelDeviceHandles to device
  /*CUDACHECK(hipMemcpyToSymbolAsync(HIP_SYMBOL(
      constSmChannels), it->second.smChannelDeviceHandles.data(),
      sizeof(mscclpp::DeviceHandle<mscclpp::SmChannel>) * it->second.smChannelDeviceHandles.size(), 0,
      hipMemcpyHostToDevice, stream));*/
  CUDACHECK(allgather((int*)sendbuff, (int*)comm->scratchBuff.get(), (int*)recvbuff,
                      rank, nRanksPerNode, nRank, bytes / sizeof(int), stream, constSmChannels));
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclSend(const void* sendbuff, size_t count, ncclDataType_t datatype, int peer, ncclComm_t comm,
                               hipStream_t stream) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclRecv(void* recvbuff, size_t count, ncclDataType_t datatype, int peer, ncclComm_t comm,
                               hipStream_t stream) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclAllToAll(const void* sendbuff, void* recvbuff, size_t count, ncclDataType_t datatype,
                                   ncclComm_t comm, hipStream_t stream) {
  // TODO: implement this function
  return ncclInternalError;
}

NCCL_API ncclResult_t ncclGroupStart() {
  // Do nothing
  return ncclSuccess;
}

NCCL_API ncclResult_t ncclGroupEnd() {
  // Do nothing
  return ncclSuccess;
}
