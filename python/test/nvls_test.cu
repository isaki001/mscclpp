#include "hip/hip_runtime.h"
// Copyright (c) Microsoft Corporation.
// Licensed under the MIT license.

#include <mscclpp/concurrency_device.hpp>
#include <mscclpp/nvls_device.hpp>
#include <mscclpp/poll_device.hpp>
#include <mscclpp/semaphore_device.hpp>

__device__ mscclpp::DeviceSyncer deviceSyncer;

#define MULTIMEM_ST(val, ptr)                                                                                   \
  asm volatile("multimem.st.global.v4.f32 [%0], {%1,%2,%3,%4};" ::"l"(ptr), "r"(val.x), "r"(val.y), "r"(val.z), \
               "r"(val.w)                                                                                       \
               : "memory");
// specific PTX for fp16 reduction. bf16 would be multimem.ld_reduce.global.add.v4.bf16x2 etc
#define MULTIMEM_LD(val, ptr)                                     \
  asm("multimem.ld_reduce.global.add.v4.f32 {%0,%1,%2,%3}, [%4];" \
      : "=r"(val.x), "=r"(val.y), "=r"(val.z), "=r"(val.w)        \
      : "l"(ptr)                                                  \
      : "memory");

extern "C" __global__ void __launch_bounds__(1024, 1)
    nvls_test(mscclpp::DeviceMulticastPointerDeviceHandle nvlsPtrs,
              mscclpp::SmDevice2DeviceSemaphoreDeviceHandle* semaphores, int my_rank, int nranks, int nbytes) {
  int nelem = nbytes / sizeof(float);
  float* dev_ptr = (float*)nvlsPtrs.devicePtr;
  float* mc_ptr = (float*)nvlsPtrs.mcPtr;
  int tid = threadIdx.x;
  int bid = blockIdx.x;

  for (int idx = bid * blockDim.x + tid; idx < nelem; idx += blockDim.x * gridDim.x) {
    dev_ptr[idx] = my_rank;
  }
  deviceSyncer.sync(gridDim.x);
  if (tid == 0 && bid == 0) {
    __threadfence_system();
  }

  if (bid == 0) {
    if (tid < nranks && tid != my_rank) {
      semaphores[tid].signal();
      semaphores[tid].wait();
    }
  }
  deviceSyncer.sync(gridDim.x);

  int my_st = ((int64_t)nelem * (int64_t)my_rank) / (int64_t)nranks;
  int my_en = ((int64_t)nelem * (int64_t)(my_rank + 1)) / (int64_t)nranks;

  int my_offset = (tid + bid * blockDim.x) * 4;
  int my_step = blockDim.x * gridDim.x * 4;

  for (int idx = my_st + my_offset; idx < my_en; idx += my_step) {
    uint4 val;
    MULTIMEM_LD(val, mc_ptr + idx);
    MULTIMEM_ST(val, mc_ptr + idx);
  }

  deviceSyncer.sync(gridDim.x);
  if (tid == 0 && bid == 0) {
    __threadfence_system();
  }

  if (bid == 0) {
    if (tid < nranks && tid != my_rank) {
      semaphores[tid].signal();
      semaphores[tid].wait();
    }
  }
  deviceSyncer.sync(gridDim.x);

  for (int idx = bid * blockDim.x + tid; idx < nelem; idx += blockDim.x * gridDim.x) {
    if (dev_ptr[idx] != ((nranks * (nranks - 1)) / 2)) {
      __assert_fail("dev_ptr[idx] != nranks", __FILE__, __LINE__, __PRETTY_FUNCTION__);
    }
  }
}
